#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <errno.h>
#include <time.h>
#include <stdbool.h>

void revArr(int* arr, int start, int end);

// Part 2 of 2: implement the fast kernel using shared memory
__global__ void reverseArrayBlock(int *d_out, int *d_in)
{
    extern _shared_ int s_data[];

    int inOffset  = blockDim.x * blockIdx.x;
    int in  = inOffset + threadIdx.x;

    // Load one element per thread from device memory and store it
    // in reversed order into temporary shared memory
    s_data[blockDim.x - 1 - threadIdx.x] = d_in[in];

    // Block until all threads in the block have written their data to shared mem
    __syncthreads();

    // write the data from shared memory in forward order,
    // but to the reversed block offset as before

    int outOffset = blockDim.x * (gridDim.x - 1 - blockIdx.x);

    int out = outOffset + threadIdx.x;
    d_out[out] = s_data[threadIdx.x];
}

////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////
int main( int argc, char** argv)
{
    // pointer for host memory and size
    int *h_a;
    int dimA = 16*1024*1024; // 256K elements (1MB total)

    // pointer for device memory
    int *d_b, *d_a;

    // define grid and block size
    int numThreadsPerBlock = 8;

    // Compute number of blocks needed based on array size and desired block size
    int numBlocks = dimA / numThreadsPerBlock;

    // Part 1 of 2: Compute the number of bytes of shared memory needed
    // This is used in the kernel invocation below
    int sharedMemSize = numThreadsPerBlock * sizeof(int);

    // allocate host and device memory
    size_t memSize = numBlocks * numThreadsPerBlock * sizeof(int);
    h_a = (int *) malloc(memSize);
    hipMalloc( (void **) &d_a, memSize );
    hipMalloc( (void **) &d_b, memSize );

    int* test_arr;
    test_arr = (int*) malloc(dimA * sizeof(int));

    // Initialize input array on host
    int val;
    srand(time(0));
    for (int i = 0; i < dimA; ++i)
    {
        val = rand();
        h_a[i] = val;
        test_arr[i] = val;
    }

    // Copy host array to device array
    hipMemcpy( d_a, h_a, memSize, hipMemcpyHostToDevice );

    // launch kernel
    dim3 dimGrid(numBlocks);
    dim3 dimBlock(numThreadsPerBlock);
    reverseArrayBlock<<< dimGrid, dimBlock, sharedMemSize >>>( d_b, d_a );

    // block until the device has completed
    hipDeviceSynchronize();

    // device to host copy
    hipMemcpy( h_a, d_b, memSize, hipMemcpyDeviceToHost );

    // Reverse test array
    revArr(test_arr, 0, dimA-1);

    // verify the data returned to the host is correct
    for (int i = 0; i < dimA; i++)
    {
        //assert(h_a[i] == dimA - 1 - i );
        assert(h_a[i] == test_arr[i]);
    }

    // free device memory
    hipFree(d_a);
    hipFree(d_b);

    // free host memory
    free(h_a);
    free(test_arr);

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    printf("Correct!\n");

    return 0;
}

void revArr(int* arr, int start, int end) {
  while (start < end) {
    int temp = arr[start];
    arr[start] = arr[end];
    arr[end] = temp;
    start++;
    end--;
  }
}

// nvcc hw10.cu -o hw10